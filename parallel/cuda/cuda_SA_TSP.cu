/*
	Simulated Annealing algorithm for Traveling Salesman Problem
	@@ CUDA version: no parallel optimization, single thread
	
	Input: xxx.tsp file
	Output: optimal value (total distance)
			& solution route: permutation of {1, 2, ..., N}
*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <algorithm>
#include <sys/time.h>
#include <pthread.h>
#include <hiprand/hiprand_kernel.h>
#define MAXITER 20		// Proposal 20 routes and then select the best one
#define THRESH1 0.1		// Threshold 1 for the strategy
#define THRESH2 0.89	// Threshold 2 for the strategy
#define RELAX 400		// The times of relaxation of the same temperature
#define ALPHA 0.999		// Cooling rate
#define INITEMP 99.0	// Initial temperature
#define STOPTEMP 0.001	// Termination temperature
#define MAXLAST 3		// Stop if the tour length keeps unchanged for MAXLAST consecutive temperature
#define MAXN 250		// only support N <= 250
#define THREADITER 200
using namespace std;

float minTourDist = -1;		// The distance of shortest path
int *minTour = NULL;		// The shortest path
int N = 0;					// Number of cities
float *dist = NULL;	// The distance matrix, use (i-1) instead of i

int *currTour = NULL;
int blockNum = 1;		// block number
int threadNum = 1;	// thread number
int globalIter = -1;	// global iteration count
pthread_mutex_t mutex = PTHREAD_MUTEX_INITIALIZER;

class rand_x { 
    unsigned int seed;
public:
    rand_x(int init) : seed(init) {}

    int operator()(int limit) {
        int divisor = RAND_MAX/(limit+1);
        int retval;

        do { 
            retval = rand_r(&seed) / divisor;
        } while (retval > limit);

        return retval;
    }        
};

/* load the data */
void loadFile(char* filename) {
	FILE *pf;

	pf = fopen(filename, "r");
	if (pf == NULL) {
		printf("Cannot open the file!\n");
		exit(1);
	}
	char buff[200];
	fscanf(pf, "NAME: %[^\n]s", buff);
	printf("%s\n", buff);
	fscanf(pf, "\nTYPE: TSP%[^\n]s", buff);
	printf("%s\n", buff);
	fscanf(pf, "\nCOMMENT: %[^\n]s", buff);
	printf("%s\n", buff);
	fscanf(pf, "\nDIMENSION: %d", &N);
	printf("The N is: %d\n", N);
	fscanf(pf, "\nEDGE_WEIGHT_TYPE: %[^\n]s", buff);
	printf("the type is: %s\n", buff);
	dist = (float *)malloc(sizeof(float) * N * N);
	memset(dist, 0, sizeof(float) * N * N);
	if (strcmp(buff, "EUC_2D") == 0) {
		fscanf(pf, "\nNODE_COORD_SECTION");
		float nodeCoord[MAXN][2] = {};
		int nid;
		float xx, yy;
		for (int i = 0; i < N; ++i) {
			fscanf(pf, "\n%d %f %f", &nid, &xx, &yy);
			nodeCoord[i][0] = xx;
			nodeCoord[i][1] = yy;
		}
		float xi, yi, xj, yj;
		for (int i = 0; i < N; ++i) {
			for (int j = i + 1; j < N; ++j) {
				xi = nodeCoord[i][0];
				yi = nodeCoord[i][1];
				xj = nodeCoord[j][0];
				yj = nodeCoord[j][1];
				dist[i*N + j] = (float)sqrt((xi - xj) * (xi - xj) + (yi - yj) * (yi - yj));
				dist[j*N + i] = dist[i*N + j];
			}
		}
	}
	else if (strcmp(buff, "EXPLICIT") == 0) {
		fscanf(pf, "\nEDGE_WEIGHT_FORMAT: %[^\n]s", buff);
		fscanf(pf, "\n%[^\n]s", buff);
		char *disps = strstr(buff, "DISPLAY_DATA_TYPE");
		if (disps != NULL) {
			fscanf(pf, "\nEDGE_WEIGHT_SECTION");
		}
		float weight;
		for (int i = 0; i < N; ++i) {
			for (int j = 0; j <= i; ++j) {
				fscanf(pf, "%f", &weight);
				dist[i*N + j] = weight;
				dist[j*N + i] = weight;
			}
		}
	}
	return;
}

/* Calculate the length of the tour */
float tourLen(int *tour) {
	if (tour == NULL) {
		printf("tour not exist!\n");
		return -1;
	}
	float cnt = 0;
	for (int i = 0; i < N - 1; ++i) {
		cnt += dist[tour[i]*N + tour[i+1]];
	}
	cnt += dist[tour[N-1]*N + tour[0]];
	return cnt;
}

/* the main simulated annealing function */
__global__ void saTSP(int cityCnt, int* globalTour, hiprandState *randStates,  float *dev_dist, float temperature, int relaxiter) {
	int thid = (blockIdx.x * blockDim.x) + threadIdx.x;
	int *tour = &globalTour[thid * cityCnt];
	float currLen = 0;
	for (int i = 0; i < cityCnt - 1; ++i) {
		currLen += dev_dist[tour[i]*cityCnt + tour[i+1]];
	}
	currLen += dev_dist[tour[cityCnt-1]*cityCnt + tour[0]];
	//float temperature = INITEMP;
	//float lastLen = currLen;
	//int contCnt = 0; // the continuous same length times
	int iterCnt = 0;
	while (temperature > STOPTEMP) {
		temperature *= ALPHA;
		iterCnt += 1;
		/* stay in the same temperature for RELAX times */
		for (int i = 0; i < relaxiter; ++i) {
			/* Proposal 1: Block Reverse between p and q */
			int p = (int)(hiprand_uniform(&(randStates[thid])) * (float)(cityCnt + 10)) % cityCnt;
			int q = (int)(hiprand_uniform(&(randStates[thid])) * (float)(cityCnt + 10)) % cityCnt;
			// If will occur error if p=0 q=N-1...
			if (abs(p - q) == cityCnt - 1) {
				p = (int)(hiprand_uniform(&(randStates[thid])) * (float)(cityCnt - 3));
				q = (int)(hiprand_uniform(&(randStates[thid])) * (float)(cityCnt - 2));
			}
			if (p == q) {
				q = (q + 2) % cityCnt;
			}
			if (p > q) {
				int tmp = p;
				p = q;
				q = tmp;
			}
			int p1 = (p - 1 + cityCnt) % cityCnt;
			int q1 = (q + 1) % cityCnt;
			int tp = tour[p], tq = tour[q], tp1 = tour[p1], tq1 = tour[q1];
			float delta = dev_dist[tp*cityCnt + tq1] + dev_dist[tp1*cityCnt + tq] - dev_dist[tp*cityCnt + tp1] - dev_dist[tq*cityCnt + tq1];

			/* whether to accept the change */
			if ((delta < 0) || ((delta > 0) && 
				(expf(-delta/temperature) > hiprand_uniform(&(randStates[thid]))))) {
				currLen = currLen + delta;
				int mid = (q - p) >> 1;
				int tmp;
				for (int k = 0; k <= mid; ++k) {
					tmp = tour[p+k];
					tour[p+k] = tour[q-k];
					tour[q-k] = tmp;
				}
				//currLen = tourLen(tour);
			}

		}
	/*
		if ((currLen - lastLen < 1e-2) && (currLen - lastLen > -1e-2)) {
			contCnt += 1;
			if (contCnt >= MAXLAST) {
				//printf("unchanged for %d times1!\n", contCnt);
				break;
			}
		}
		else
			contCnt = 0;
		lastLen = currLen;
	*/
	}
	
	return;
}

__global__ void setup_kernel_randomness(hiprandState * state, unsigned long seed)
{
	int s_id = (blockIdx.x*blockDim.x) + threadIdx.x;
	hiprand_init(seed*s_id, s_id, 0, &state[s_id]);
}

int main(int argc, char **argv) {
	hipError_t err = hipSuccess;
	float *dev_dist;
	
	if (argc < 2) {
		printf("Usage: ./cuda_tsp <filename> <blockNum> <threadNum>\n");
		return 0;
	}
	else {
		loadFile(argv[1]);
		err = hipMalloc((void **)&dev_dist, sizeof(float) * N * N);
		if (err != hipSuccess) {
			fprintf(stderr, "cudaMalloc() failed\n");
			exit(1);
		}

		hipMemcpy((void *)dev_dist, dist, sizeof(float) * N * N, hipMemcpyHostToDevice);
	}
	if (argc == 4) {
		blockNum = atoi(argv[2]);
		threadNum = atoi(argv[3]);
	}
	printf("blockNum is: %d, threadNum is: %d\n", blockNum, threadNum);
	struct timeval start, stop;
	gettimeofday(&start, NULL);
	srandom(time(0));
	int *dev_currTour; // currTour on device;
	int itersCnt = blockNum * threadNum; // total iterations
	err = hipMalloc((void **)&dev_currTour, sizeof(int)*N*itersCnt);
	if (err != hipSuccess) {
		fprintf(stderr, "cudaMalloc() failed\n");
		exit(1);
	}

	srand(time(0));
	currTour = (int *)malloc(sizeof(int) * N * itersCnt);
	for (int i = 0; i < itersCnt; ++i) {
		for (int j = 0; j < N; ++j) {
			currTour[i*N + j] = j;
		}
		random_shuffle(currTour+i*N, currTour+(i+1)*N);
		/*for (int j = 0; j < N; ++j) {
			printf("%d ", currTour[i*N + j]);
		}
		printf("%d before: %f\n", i, tourLen(currTour + i*N));*/
	}
	err = hipMemcpy(dev_currTour, currTour, itersCnt * N * sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "cudaMalloc() for dev_currTour failed\n");
		exit(1);
	}

	// allocate random seed for each thread
	hiprandState *devStates;
	hipMalloc((void **)&devStates, itersCnt * sizeof(hiprandState));	
	setup_kernel_randomness<<<blockNum, threadNum>>>(devStates, time(0));
	hipDeviceSynchronize();

	float currLen = 0;
	
	float temperature = INITEMP;
	int contCnt = 0;
	float tempstep = pow(ALPHA, THREADITER);
	//while (temperature > STOPTEMP) {
		//printf("%.06f \n", temperature);
		saTSP<<<blockNum, threadNum>>>(N, dev_currTour, devStates, dev_dist, temperature, RELAX);
		hipDeviceSynchronize();	
	//	temperature *= tempstep;
	//}

	minTour = (int *)malloc(sizeof(int) * N);
	memset(currTour, 0, itersCnt * N * sizeof(int));
	err = hipMemcpy(currTour, dev_currTour, itersCnt * N * sizeof(int), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "cudaMemcpyc(Device to Host) failed with %d\n", err);
		exit(1);
	}

	/* find the minimal answer */
	int minidx = 0;
	for (int i = 0; i < itersCnt; ++i) {
		currLen = tourLen(&currTour[i * N]);
		/*for (int j = 0; j < N; ++j) {
			printf("%d ", currTour[i*N + j]);
		}

		printf("%d after: %f\n", i, currLen);*/
		if ((currLen < minTourDist) || (minTourDist < 0)) {
			minTourDist = currLen;
			minidx = i;
		}
	}
	for (int i = 0; i < N; ++i) {
		minTour[i] = currTour[minidx * N + i];
	}
	gettimeofday(&stop, NULL);

	// ------------- Print the result! -----------------
	int tottime = stop.tv_sec - start.tv_sec;
	int timemin = tottime / 60;
	int timesec = tottime % 60;
	printf("Total time usage: %d min %d sec. \n", timemin, timesec);
	printf("N is %d, The shortest length is: %f\n And the tour is: \n", N, minTourDist);
	for (int i = 0; i < N; ++i) {
		printf("%d \n", minTour[i]+1);
	}
	free(dist);
	free(minTour);
	free(currTour);
	
	return 0;
}
